#include "hip/hip_runtime.h"
#include <mpi.h>
#include "mk5mode_gpu.cuh"
#include "gpumode_kernels.cuh"
#include "gpudecode.cuh"
//#include "mk5.h"
#include "alert.h"
#include <iostream>
#include <bitset>
#include <unistd.h>

#define NOT_SUPPORTED(x) { std::cerr << "Whoops, we don't support this on the GPU: " << x << std::endl; exit(1); }

Mk5_GPUMode::Mk5_GPUMode(Configuration * conf, int confindex, int dsindex, int recordedbandchan, int chanstoavg, int bpersend, int gsamples, int nrecordedfreqs, double recordedbw, double * recordedfreqclkoffs, double * recordedfreqclkoffsdelta, double * recordedfreqphaseoffs, double * recordedfreqlooffs, int nrecordedbands, int nzoombands, int nbits, Configuration::datasampling sampling, Configuration::complextype tcomplex, bool fbank, bool linear2circular, int fringerotorder, int arraystridelen, bool cacorrs, int framebytes, int framesamples, Configuration::dataformat format)
  : GPUMode(conf, confindex, dsindex, recordedbandchan, chanstoavg, bpersend, gsamples, nrecordedfreqs, recordedbw, recordedfreqclkoffs, recordedfreqclkoffsdelta, recordedfreqphaseoffs, recordedfreqlooffs, nrecordedbands, nzoombands, nbits, sampling, tcomplex, recordedbandchan*2+4, fbank, linear2circular, fringerotorder, arraystridelen, cacorrs, recordedbw*2)
{
  char formatname[64];

  fanout = config->genMk5FormatName(format, nrecordedbands, recordedbw, nbits, sampling, framebytes, conf->getDDecimationFactor(confindex, dsindex), config->getDAlignmentSeconds(confindex, dsindex), conf->getDNumMuxThreads(confindex, dsindex), formatname);
  invalid = 0;

  if(fanout < 0)
    initok = false;
  else
  {
    // since we allocated the max amount of space needed above, we need to change
    // this to the number actually needed.
    this->framesamples = framesamples;
    if (usecomplex) {
      unpacksamples = recordedbandchan;
      samplestounpack = recordedbandchan;
    } else {
      unpacksamples = recordedbandchan*2;
      samplestounpack = recordedbandchan*2;
    }
    //create the mark5_stream used for unpacking
    mark5stream = new_mark5_stream( new_mark5_stream_unpacker(0), new_mark5_format_generic_from_string(formatname) );
    if(mark5stream == 0)
    {
      cfatal << startl << "Mk5_GPUMode::Mk5_GPUMode : mark5stream is null" << endl;
      initok = false;
    }
    else
    {
      if(conf->isNetwork(dsindex))
        mark5stream->blanker = blanker_none;
      if(mark5stream->samplegranularity > 1)
        samplestounpack += mark5stream->samplegranularity;
      string orig_streamname(mark5stream->streamname);
      sprintf(mark5stream->streamname, "DS%d <%s>", dsindex, orig_streamname.c_str());
      if(framesamples != mark5stream->framesamples)
      {
        cfatal << startl << "Mk5_GPUMode::Mk5_GPUMode : framesamples inconsistent (told " << framesamples << "/ stream says " << mark5stream->framesamples << ") - for stream index " << dsindex << endl;
        initok = false;
      }
      else
      {
        this->framesamples = mark5stream->framesamples;
      }
      if(format == Configuration::INTERLACEDVDIF)
      {
        invalid = new int[nrecordedbands];
        perbandweights = new f32*[config->getNumBufferedFFTs(configindex)];
        for(int i=0;i<config->getNumBufferedFFTs(configindex);++i)
        {
          perbandweights[i] = new f32[nrecordedbands];
          for(int b = 0; b < nrecordedbands; ++b)
          {
            perbandweights[i][b] = 0.0;
          }
        }
      }
    }
  }
}

Mk5_GPUMode::~Mk5_GPUMode()
{
  delete_mark5_stream(mark5stream);
  if(invalid)
  {
    delete [] invalid;
  }
}

float Mk5_GPUMode::unpack(int sampleoffset, int subloopindex)
{
  float goodsamples = 0;
  int mungedoffset = 0;

  //work out where to start from
  unpackstartsamples = sampleoffset - (sampleoffset % mark5stream->samplegranularity);

  //unpack one frame plus one FFT size worth of samples
  if(usecomplex) 
  {
    NOT_SUPPORTED("unpack - usecomplex");
  }
  if(mark5stream->samplegranularity > 1)
    { // CHRIS not sure what this is mean to do
      // WALTER: unpacking of some mark5 modes (those with granularity > 1) must be unpacked not as individual samples but in groups of sample granularity
    int erasedsamples = 0;

    mungedoffset = sampleoffset % mark5stream->samplegranularity;
    for(int i = 0; i < mungedoffset; i++) {
      for(int b = subloopindex * numrecordedbands; b < subloopindex * numrecordedbands + mark5stream->nchan; ++b) {
        if(unpackedarrays_gpu->ptr()[b][i] != 0.0) {
            unpackedarrays_gpu->ptr()[b][i] = 0.0;
          erasedsamples++;
        }
      }
    }
    for(int i = unpacksamples + mungedoffset; i < samplestounpack; i++) {
      for(int b = subloopindex * numrecordedbands; b < subloopindex * numrecordedbands + mark5stream->nchan; ++b) {
        if(unpackedarrays_gpu->ptr()[b][i] != 0.0) {
            unpackedarrays_gpu->ptr()[b][i] = 0.0;
          erasedsamples++;
        }
      }
    }
    goodsamples -= erasedsamples/(float)(mark5stream->nchan);
  }
  if(perbandweights)
  {
      if(usecomplex)
      {
          NOT_SUPPORTED("unpack - usecomplex");
      }
      else
      {
          blank_vdif_EDV4(data, unpackstartsamples, &unpackedarrays_gpu->ptr()[subloopindex * numrecordedbands], samplestounpack, invalid);
      }

      int totalinvalid = 0;
      for(int b = 0; b < mark5stream->nchan; ++b)
      {
          perbandweights[subloopindex][b] = (goodsamples - invalid[b])/(float)unpacksamples;
          totalinvalid += invalid[b];
      }

      goodsamples -= (float)totalinvalid/(float)(mark5stream->nchan);
  }

  if(goodsamples < 0)
  {
    cerror << startl << "Error trying to unpack Mark5 format data at sampleoffset " << sampleoffset << " from data seconds " << datasec << " plus " << datans << " ns!!!" << endl;
    goodsamples = 0;
    for(int b = 0; b < mark5stream->nchan; ++b)
      invalid[b] = 0;
  }

  return goodsamples/(float)unpacksamples;
}

// This unpack_all call (like all GPUMode unpack_alls) unpacks everything
// (except the header of course), including the unnecessary period between the
// start of the frame and the first sample in the observation. (We do this to
// avoid a corner-case where unpacking has to begin inside a word, or worse,
// inside a byte)
void Mk5_GPUMode::unpack_all(int framestounpack, int frame_size) {
    // Hacky little workaround to get the stream struct back !! May not be needed !!
  /*
    mark5_stream *tmp_mk5stream;
    hipMallocManaged(&tmp_mk5stream, sizeof(mark5_stream));
    *tmp_mk5stream = *mark5stream;
    */

    std::cout << "frames to unpack: " << framestounpack << std::endl;

    const int BYTES_PER_WORD = 4;
    const int unpack_threads = 256;
    const int total_bytes = framestounpack * frame_size;
    assert(total_bytes % BYTES_PER_WORD == 0); // VDIF must always be an integer number of words
    const int total_words = total_bytes / BYTES_PER_WORD;
    const int blocks = (total_bytes + unpack_threads - 1) / unpack_threads;

    const int HEADERLENGTH_BYTES = 4 * 8;
    const int payloadlength_bytes = frame_size - HEADERLENGTH_BYTES;
    const int payloadlength_words = payloadlength_bytes / BYTES_PER_WORD;

    std::cout << " cuda blocks: " << blocks << " ; total_bytes: " << total_bytes << "; total_words: " << total_words << std::endl;
    gpu_unpack<<<blocks, unpack_threads, 0, cuStream>>>(
        (char*)packeddata_gpu->gpuPtr(),
        unpackedarrays_gpu->gpuPtr(),
        valid_frames->gpuPtr(),
        payloadlength_words,
        frame_size,
        total_words
      );

    /*
    int unpack_blocks = (framestounpack + unpack_threads - 1) / unpack_threads;

    hipStreamSynchronize(cuStream);
    std::cout << "call gpu_unpack - unpack_blocks = " << unpack_blocks << "; unpack_threads = " << unpack_threads << std::endl;
    gpu_unpack<<<unpack_blocks, unpack_threads, 0, cuStream>>>(tmp_mk5stream, packeddata_gpu->gpuPtr(), unpackedarrays_gpu->gpuPtr(), framestounpack, valid_frames->gpuPtr());
    */

    /*
    hipMemset(valid_frames->gpuPtr(), 1, framestounpack);

    // Unfortunately we have to block here since we need the valid frames to find the correct dataweights
    valid_frames->sync();
    valid_frames->copyToHost();
    valid_frames->sync();
    */

    //hipFree(tmp_mk5stream);

}
// vim: shiftwidth=2:softtabstop=2:expandtab
